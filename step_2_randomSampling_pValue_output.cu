
/* Execution Format : ./<exe> <drug_result_1_dict_compounds.txt> <drug_result_2_dict_compounds.txt> <drug_result_1_dict_proteins.txt> <drug_result_2_dict_proteins.txt> <para.txt> <drug name>
*/

#include <stdio.h>
#include <errno.h>
#include <math.h>
#include <string.h>
#include <unistd.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/dir.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

//Structure for key-value pairs in dictionary
struct kvpair {
    char *key;
    int value;
    struct kvpair *next;
};

//Structure for dictionary
typedef struct dictionary {
    int size;           // size of the pointer table 
    int n;              // number of elements stored 
    struct kvpair **table;
}*Dictionary;

//Structure for nodes in the CUDA hashtable
typedef struct node {
	char key[80];
	int index;
	struct node *next;
} Node;

//Function to compute hash value
unsigned long computeHash(const char *s)
{
    unsigned const char *us;
    unsigned long h;
    h = 0;
    for(us = (unsigned const char *) s; *us; us++) {
        h = h * 401 + *us;
    }
    return h;
}

// Function to create empty dictionary
Dictionary createDictionary()
{
    Dictionary d;
    int i;
    d = (Dictionary)malloc(sizeof(*d));
	if(d==NULL){
		printf("dictionary d malloc failed\n");
		exit(0);
	}
    assert(d != 0);
    d->size = 102397;
    d->n = 0;
    d->table = (kvpair **)malloc(sizeof(struct kvpair *) * d->size);
	if(d->table==NULL){
		printf("d->table malloc failed\n");
		exit(0);
	}
    assert(d->table != 0);
    for(i = 0; i < d->size; i++) 
    	d->table[i] = 0;
    return d;
}

// Function to insert a new key-value pair into the dictionary 
void insertDictionary(Dictionary d, const char *key, int val)
{
    struct kvpair *e;
    unsigned long h;
    assert(key);
    assert(val);
    e = (kvpair*)malloc(sizeof(*e));
	if(e==NULL){
		printf("e kvpair malloc failed\n");
		exit(0);
	}
    assert(e);
    e->key = strdup(key);
    e->value = val;
    h = computeHash(key) % d->size;
    e->next = d->table[h];
    d->table[h] = e;
    d->n++;
	return;
}

//Function to search for a key in the dictionary, returns NULL or the Node of the key if found in the dictionary
struct kvpair * searchDictionary(Dictionary d, const char *key)
{
    struct kvpair *e;
    for(e = d->table[computeHash(key) % d->size]; e != 0; e = e->next) {
        if(!strcmp(e->key, key)) {
            return e;
        }
    }
    return NULL;
}

// Function to delete key-value pair in dictionary --this is currently not used
void deleteDictionary(Dictionary d, const char *key)
{
    struct kvpair **prev;          
    struct kvpair *e;              
    for(prev = &(d->table[computeHash(key) % d->size]); 
        *prev != 0; 
        prev = &((*prev)->next)) {
        if(!strcmp((*prev)->key, key)) {
            e = *prev;
            *prev = e->next;
            free(e->key);
            free(e);
            return;
        }
    }
}

//Function to free dictionary
void destroyDictionary(Dictionary d)
{
    int i;
    struct kvpair *e;
    struct kvpair *next;
    for(i = 0; i < d->size; i++) {
        for(e = d->table[i]; e != 0; e = next) {
            next = e->next;
            free(e->key);
            free(e);
        }
    }
    free(d->table);
    free(d);
    return;
}

//Function to compute normal distribution of a value, equivalent to Python's CDF.norm from NVIDIA CUDA samples
__device__ float CND(float d)
{
    const double       A1 = 0.31938153;
    const double       A2 = -0.356563782;
    const double       A3 = 1.781477937;
    const double       A4 = -1.821255978;
    const double       A5 = 1.330274429;
    const double RSQRT2PI = 0.39894228040143267793994605993438;
    double
    K = 1.0 / (1.0 + 0.2316419 * fabs(d));
    double
    cnd = RSQRT2PI * exp(- 0.5 * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));
    if (d > 0)
        cnd = 1.0 - cnd;
    return cnd;
}

//Function to remove specific characters from input string, used to remove carriage returns
void removeChar(char *str, char garbage) {
    char *src, *dst;
    for (src = dst = str; *src != '\0'; src++) {
        *dst = *src;
        if (*dst != garbage) dst++;
    }
    *dst = '\0';
	return;
}

//Function used by qsort to sort the records based on number of tokens
int sort(const void* a, const void* b)
{

	 char *ia = strdup(*(const char **)a);
     char *ib = strdup(*(const char **)b);
	 char *split1, *saveptr, *saveptr1;
	 split1 = strtok_r(ia, ";", &saveptr);
	 split1 = strtok_r(NULL, ";", &saveptr);
	 int x = atoi(split1);
	 split1 = strtok_r(ib, ";", &saveptr1);
	 split1 = strtok_r(NULL, ";", &saveptr1);
	 return (x-atoi(split1));
}

/*Kernel function performs random sampling 
It is designed in this way: One block does one sampling and every thread processes one record. In case number of records to be processed exceed 1024, then some threads will take more than one stride. That is, some threads process more than one record during one sampling.
Stages in kernel function are: 
1) Build the dictionary 's_hashtab' is shared memory, for O(1) time lookup of keyword while sampling, first thread will ensure all the keys 'd_r1_dict_keys' are linked in the hashtable.
2) Generate the random numbers and sort them. Every thread will generate a random number and first thread in every block will sort the random numbers using iterative quick sort function. In case number of random numbers required are higher than 1024, then some threads will take more strides to generate the required number of random numbers. 
3) Shared memory initialization for sampling. This is required, as during sampling if keyword is found then we increment the count.
4) Sampling, every block performs one sampling. And, every thread will process atleast one record. That is, thread will extract the keywords/tokens in the record and then finds for the keyword in the dictionary, if found then shared memory is incremented.
5) Copy data to global memory from shared memory for Z-score and P-value calculation.

Note: Shared memory s_hashtab - is the hash table in the shared memory. 
	  Shared memory s_r1_value_list - is the value list in the shared memory.

Arguments passed to kernel function:
	  * d_r2_str - is the list of records to be used for sampling.
	  * d_r1_dict_value - Global values for vector produced from sampling.
	  * sampleTimes - number of samples.
	  * sampleSize - size of the sample.
	  * randomRange - maximum value of each random number.
	  * r1_dict_cnt - number of keywords in dictionary 1.
	  * d_r1_dict_keys - keywords of dictionary 1, to populate hashtable in kernel function.
	  * d_hashtab - global memory hash table.
	  * sampleStrides - maximum number of strides every thread will take for sampling.
	  * threadCount - number of threads per block.
	  * samplesCompleted - number of samplings completed before this kernel launch.
	  * relaunch - to decide whether kernel is launched for the first time or relaunched.
*/
__global__ void deviceDDI(char * d_r2_str, int * d_r1_dict_value, int sampleTimes, int sampleSize, int randomRange, int r1_dict_cnt, Node *d_r1_dict_keys, Node **d_hashtab, int sampleStrides, int threadCount, int samplesCompleted, bool relaunch){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j, k, x = 0, ind1, random;
	unsigned hashval;
 	char str_split[80];
	extern __shared__ int s_r1_value_list[];
    __shared__ Node *s_hashtab[5003];
	//s_r1_value_list array has array for keywords found during sampling.
	//This is used for incrementing array of keywords for sampling.
	//Build the dictionary
	if(threadIdx.x==0){
		if(!relaunch){
			//Initialize the hash table
			for(j=0;j<5003;j++){
            	s_hashtab[j] = 0;
			}
			j=0;
			k=0;
			//Build the dictionary
			for(j=0;j<r1_dict_cnt;j++){
				x=0;
				hashval = 0;
				while(d_r1_dict_keys[j].key[x]!='\0'){
					hashval = ((int)d_r1_dict_keys[j].key[x++] + 401*hashval)%5003;
				}
				d_r1_dict_keys[j].next = (s_hashtab[hashval]==0?NULL:s_hashtab[hashval]);
				s_hashtab[hashval] = &d_r1_dict_keys[j];
			}
 			for(j=0;j<5003;j++){
            	d_hashtab[j] = s_hashtab[j];
			}
		}
		else{
			for(j=0;j<5003;j++){
            	s_hashtab[j] = d_hashtab[j];
			}
		}
		//Initialize the shared memory
		for(j=0;j<(r1_dict_cnt);j++){
 			s_r1_value_list[j] = 0;
 		}
	}	
	__syncthreads();		
	//Generate the random numbers
	hiprandState_t state;
 	hiprand_init(clock64(), i, 0, &state);		
 	//Sampling
 	for(j=0;(j<sampleStrides)&&((threadIdx.x+(j*threadCount))<sampleSize);j++){
 		//char str_split[80];
 		x=0;
 		hashval = 0;
		random = hiprand(&state)%randomRange;
		for(k=random*1000;k<random*1000+1000;k++){
			if(d_r2_str[k] == '^')
				break;
 			if(d_r2_str[k] != '~'){
 		 		str_split[x++] = d_r2_str[k];
 		 		hashval = ((int)d_r2_str[k] + 401*hashval)%5003;
 			}
 			else{
 				//str_split[x] = '\0';
				Node *np = s_hashtab[hashval];
				//np = s_hashtab[hashval];
 				while((np!= NULL)&&(np!=0)){
 		 			ind1 = 0;
 		 			while((np->key[ind1] != '\0')&&(ind1<x)){
 		 				if(np->key[ind1] == str_split[ind1])
 		 					ind1++;
 		 				else
 		 					break;
 		 			}
 		 			if((np->key[ind1] == '\0')&&(ind1==x)){
						atomicAdd(&s_r1_value_list[(np->index)],1);
 		 				break;
 		 			}
 		 			if(np->next == NULL||np->next==0)
 		 				break;
 		 			np = np->next;
 				}
 				x=0;
 				hashval = 0;
 			}
 		}
 	}
	__syncthreads(); 

	//Copy to global memory from shared memory
	if(threadIdx.x==0){
		for(j=0;j<(r1_dict_cnt);j++){
 			d_r1_dict_value[(j)*(sampleTimes+1)+blockIdx.x+1+samplesCompleted] = s_r1_value_list[j];
 		}
	}
}

/*	deviceZP: Function to calculate P-value
	One thread operates on one array to compute Z-score and P-value

	Arguments passed to the kernel function:
		* d_r1_dict_value -  array holding the 
		* d_z_score - array to hold Z-scores.
	  	* d_p_value - array to hold P-values.
*/
__global__ void deviceZP(int * d_r1_dict_value, int sampleTimes,int r1_dict_cnt, float * d_z_score, float * d_p_value){
    int x, j,  i = blockDim.x * blockIdx.x + threadIdx.x;
    float mean =0,sd;
    if(i<r1_dict_cnt){
            x = 0;
            sd = 0;
            for(j=1;j<=sampleTimes;j++){
                x += d_r1_dict_value[(i*(sampleTimes+1))+j];
            }
            mean = x/(sampleTimes);
            for(j=1;j<=sampleTimes;j++){
                sd += (d_r1_dict_value[(i*(sampleTimes+1))+j]-mean)*(d_r1_dict_value[(i*(sampleTimes+1))+j]-mean);
            }
            sd = sqrt(sd/(sampleTimes));

            if(fabs(sd)>pow(10.0,-7))
                d_z_score[i] = (d_r1_dict_value[i*(sampleTimes+1)] - mean)/sd;
            else{
                if(d_r1_dict_value[i*(sampleTimes+1)] != (int)mean)
                    d_z_score[i] = d_r1_dict_value[i*(sampleTimes+1)]*100;
                else
                    d_z_score[i] = -100;
            }
            d_p_value[i] = 1-CND(d_z_score[i]);
    }
}

//Function to partition records while sorting based on Z-score, called by quickSort
int partition( float a[], int index[], int l, int r) {
   int i, j, t;
   float temp;
   float pivot = a[l];
   i = l; 
   j = r+1;		
   while( 1)
   {
   	do ++i; 
   	while( a[i] >= pivot && i <= r );
   	do --j; 
   	while( a[j] < pivot );
   	if( i >= j ) 
   		break;
   	temp = a[i]; 
   	a[i] = a[j]; 
   	a[j] = temp;
   	t = index[i];
   	index[i] = index[j];
   	index[j] = t;
   }
   temp = a[l]; 
   a[l] = a[j]; 
   a[j] = temp;
   t = index[l];
   index[l] = index[j];
   index[j] = t;
   return j;
}

//Function to quicksort the records based on Z-score
void quickSort(float a[], int index[], int l, int r)
{
   int j;
   if( l < r ) 
   {
       j = partition( a, index, l, r);
       quickSort( a, index, l, j-1);
       quickSort( a, index, j+1, r);
   }	
   return;
}


int main(int argc, char *argv[])
{
	if(argc!=7){
		printf("\nIncorrect arguments passed, Please pass <Compounds with interactions>, <Compounds without interactions>, <Proteins with interactions>, <Proteins without interactions>, <PMID Substances>, <para.txt>, <Drug Name> as arguments\n");
		exit(1);
	}
	FILE *inp_r1, *inp_r2, *inp_para, *op1, *op2;
	char *split0,*split1, *saveptr, *saveptr1, *saveptr2, *inp2_list[100000];
	char filename1[100], filename2[100], cutoffstr[20], pvaluestr[20], str1[10000], rmode[2] = "r";
	size_t len = 0;
	Dictionary d_cinp1;
	int cutoff, sampleTimes, i=0, j=0, k=0, r1_cnt, r2_cnt, r1_dict_cnt, threadCount, sampleStrides;
	float p_value, elapsedTime, totalTime=0;
	hipEvent_t start, stop;

	printf("Drug name = %s\n",argv[6]);
	printf("Read input files\n");
	d_cinp1 = createDictionary();			

	//Read the parameters from para.txt - 4th argument
	inp_para = fopen(argv[5],rmode);
	if (inp_para == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[5]);
		exit(1);
	}	
	while(1)
	{	
		fscanf(inp_para,"%[^\n]%*c", str1);
		if(feof(inp_para)) break;
		split0 = strtok_r(str1, "\t", &saveptr);
		split1 = strtok_r(NULL, "\t", &saveptr);
	 	removeChar(split0,'\r');
		removeChar(split1,'\r');
		if( strcmp(split0,"sampleTimes") == 0)
		{
			char temp[20];
			strcpy(temp, split1);
			sampleTimes = atoi(temp);	
		}
		
		 else if( strcmp(split0,"cutoff") == 0)
		{
			char temp[20];
			strcpy(temp, split1);
			strcpy(cutoffstr,temp);
			cutoff = atoi(temp);	
		}
		else if( strcmp(split0,"p_value") == 0)
		{
			char temp[20];
			strcpy(temp, split1);
			strcpy(pvaluestr,temp);
			p_value = atof(temp);	
		}
	}
	fclose(inp_para);
	printf("Number of Samples = %d\n",sampleTimes);
	if(sampleTimes <=0){
		printf("Incorrect number of samples specified = %d, value of atleast 1 is expected\n", sampleTimes);
		exit(0);
	}
	// Reading the dictionary of compounds of result 1 - 1st argument
	// Create and populate dictionary 'd_cinp1' while reading the records
	inp_r1 = fopen(argv[1], rmode);
	if (inp_r1 == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[1]);
		exit(1);
	}	
	
	r1_cnt = 0;
	r1_dict_cnt = 0;

	while(1){
		fscanf(inp_r1, "%[^\n]%*c", str1);
		if( feof(inp_r1)) break;
		removeChar(str1,'\r');
		r1_cnt++;
		len = strlen(str1);
		for(i=0;(i<len);i++){
			char *newstr = (char*)malloc(len+1);
			if(newstr==NULL){
				printf("malloc to newstr failed\n");
				exit(0);
			}
			j=0;
			while(str1[i] != '~'){
				newstr[j++] = str1[i++];
			}	
			newstr[j] = '\0';
			struct kvpair * e = searchDictionary(d_cinp1,newstr);
			if(e!=NULL){
				e->value++;
			}
			else{
				insertDictionary(d_cinp1,newstr,1);
				r1_dict_cnt++;
			}
			free(newstr);
		}
	}	
	fclose(inp_r1);

	// Reading the list of result 2- 2nd argument
	inp_r2 = fopen(argv[2], rmode);
	if (inp_r2 == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[2]);
		exit(1);
	}	
	r2_cnt = 0;
	while (1) 
	{
		fscanf(inp_r2, "%[^\n]%*c", str1);
		if( feof(inp_r2)) break;
		removeChar(str1,'\r');
		inp2_list[r2_cnt] = (char*)malloc(strlen(str1)+1);
		if(inp2_list[r2_cnt]==NULL){
			printf("malloc to inp2_list[r2_cnt] failed\n");
			exit(0);
		}
		strcpy(inp2_list[r2_cnt++],str1);		
	}
	fclose(inp_r2);
	printf("Input files read completed\n");
	printf("Sample size = %d\n", r1_cnt);
	//Sort inp2_list based on the number of tokens or length
	qsort(inp2_list,r2_cnt,sizeof(char *), sort);

	printf("Pre-process records for kernel launch\n");
	
	hipSetDevice(0);
	//populate value list for dictionary 1
	hipError_t err = hipSuccess;
	int * r1_dict_value;
	//pinned memory for optimized usage of memory transfer bandwidth
	err = hipHostMalloc((void**)&r1_dict_value, sizeof(int)*r1_dict_cnt*(sampleTimes+1), hipHostMallocDefault);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate r1_dict_value host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Device value list for dictionary 1
	int *d_r1_dict_value;
	Node * r1_dict_keys = (Node*)malloc(sizeof(Node)*r1_dict_cnt);
	if(r1_dict_keys==NULL){
		printf("malloc to r1_dict_keys failed\n");
		exit(0);
	}
	j=0;
	k=0;
	for(i=0;i<d_cinp1->size;i++)
    {
    	if(d_cinp1->table[i]!=0){
			while(1)
			{
				strcpy(r1_dict_keys[j].key,d_cinp1->table[i]->key);
				r1_dict_keys[j].index = j;
				r1_dict_keys[j].next = NULL;
				r1_dict_value[j*(sampleTimes+1)] = d_cinp1->table[i]->value;
				j++;
				if(d_cinp1->table[i]->next!= NULL)
					d_cinp1->table[i] =  d_cinp1->table[i]->next;
				else
					break;
			}
		}
	}
	destroyDictionary(d_cinp1);

	//Strip off the number of tokens from every record in list 2.
	for(i=0;i<r2_cnt;i++){
		split0 = strtok_r(inp2_list[i], ";", &saveptr1);
	}

	//Process the records for shipping to kernel
	char * temp1 = (char*) malloc(1000*r2_cnt*sizeof(char));
	if(temp1==NULL){
		printf("temp1 malloc failed\n");
		exit(0);
	}
	char * d_r2_str;
	j=0;
	for(i=0;i<r2_cnt;i++){
		for(k=0;k<1000;k++){
			if(k<strlen(inp2_list[i])){
				temp1[j++] = inp2_list[i][k];
			}
			else
				temp1[j++] = '^';
		}
		free(inp2_list[i]);
	}

	//char* d_r1_dict_list;
	//Allocate global memory for dictionary 1 keywords
	/*err = cudaMalloc((void **)&d_r1_dict_list,80*sizeof(char)*r1_dict_cnt);
	if(err != cudaSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_list (error code %s) !\n", cudaGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = cudaMemcpy(d_r1_dict_list,r1_dict_list,80*sizeof(char)*r1_dict_cnt,cudaMemcpyHostToDevice);
	if(err != cudaSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_list (error code %s) !\n", cudaGetErrorString(err));
		exit(EXIT_FAILURE);	
	}*/
	
	//Allocate global memory for input list 2 records
	err = hipMalloc((void **)&d_r2_str,1000*sizeof(char)*r2_cnt);
	if(err != hipSuccess){
			fprintf(stderr,"Failed to allocate device d_r2_str (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r2_str,temp1,1000*sizeof(char)*r2_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r2_str (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Allocate global memory for dictionary 1 value list 
	err = hipMalloc((void **)&d_r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r1_dict_value,r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	Node * d_r1_dict_keys = NULL;
	err = hipMalloc((void **)&d_r1_dict_keys,sizeof(Node)*r1_dict_cnt);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	err = hipMemcpy(d_r1_dict_keys,r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	Node **hashtab;
	Node **d_hashtab;
	err = hipMalloc((void **)&d_hashtab,sizeof(Node*)*5003);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_hashtab (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	hashtab = (Node**)malloc(sizeof(Node*)*5003);
	if(hashtab==NULL){
		printf("hashtab malloc failed\n");
		exit(0);
	}
	printf("Sampling for compounds begin\n");
	for(i=0;i<sampleTimes;i=i+256){

		threadCount = (r1_cnt>1024)?1024:r1_cnt;
		//sampleStrides: maximum number of strides every thread needs to take for sampling
		sampleStrides = ceil(r1_cnt/threadCount);
		printf("Kernel deviceDDI launched with %d blocks of %d threads each\n", (sampleTimes-i)>256?256:(sampleTimes-i), threadCount);

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord( start, 0 );

		deviceDDI<<<(sampleTimes-i)>256?256:(sampleTimes-i), threadCount, (sizeof(int)*(r1_dict_cnt))>>>(d_r2_str, d_r1_dict_value, sampleTimes, r1_cnt, r2_cnt, r1_dict_cnt, d_r1_dict_keys, d_hashtab, sampleStrides, threadCount,i,(i==0)?false:true);
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &elapsedTime, start, stop );
		hipEventDestroy( start );
		hipEventDestroy( stop );
		err = hipDeviceSynchronize();
		if(err != hipSuccess){
			fprintf(stderr,"Failed to launch deviceDDI kernel device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}
		err = hipGetLastError();
		if(err != hipSuccess){
			fprintf(stderr,"Failed to launch deviceDDI kernel device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}

		err = hipMemcpy(r1_dict_value,d_r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyDeviceToHost);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_value from device to Host(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}

		err = hipMemcpy(d_r1_dict_value, r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_value from host to device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}
		
		err = hipMemcpy(r1_dict_keys,d_r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyDeviceToHost);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_keys to host (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(d_r1_dict_keys,r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(hashtab,d_hashtab,sizeof(Node*)*5003,hipMemcpyDeviceToHost);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_hashtab to host (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(d_hashtab,hashtab,sizeof(Node*)*5003,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_hashtab (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(d_r2_str,temp1,1000*sizeof(char)*r2_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r2_str (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

	}
	printf("Sampling for compounds completed\n");

    err = hipFree(d_r2_str);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r2_str (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipFree(d_r1_dict_keys);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
    free(temp1);

    float *d_z_score;
    float *z_score_arr;
    //Allocate array for Z-score, pinned memory for optimized usage of memory transfer bandwidth
    err = hipHostMalloc((void**)&z_score_arr, sizeof(float)*r1_dict_cnt, hipHostMallocDefault);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate z-score host (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipMalloc((void **)&d_z_score,sizeof(float)*r1_dict_cnt);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate device d_z_score (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    float *d_p_value;
    float *p_value_arr;
    //Allocate array for P-value, pinned memory for optimized usage of memory transfer bandwidth
    err = hipHostMalloc((void**)&p_value_arr, sizeof(float)*r1_dict_cnt, hipHostMallocDefault);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate p-value host (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipMalloc((void **)&d_p_value,sizeof(float)*r1_dict_cnt);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate device d_p_value (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
	
	printf("Z-score and P-value calculation for Compounds begin\n");
	printf("Kernel deviceZP launched with %d blocks of %d threads each\n", (int)ceil(r1_dict_cnt/256.0), 256);
    totalTime += elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    deviceZP<<<ceil(r1_dict_cnt/256.0), 256>>>(d_r1_dict_value, sampleTimes, r1_dict_cnt, d_z_score, d_p_value);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &elapsedTime, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	err = hipDeviceSynchronize();
	if(err != hipSuccess){
		fprintf(stderr,"Failed to launch deviceZP kernel device(error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE); 
	}
	err = hipGetLastError();
	if(err != hipSuccess){
		fprintf(stderr,"Failed to launch deviceZP kernel device(error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	printf("Z-score, P-value calculation completed\n");
	err = hipMemcpy(z_score_arr,d_z_score,sizeof(float)*r1_dict_cnt,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy from z-score device to host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(p_value_arr,d_p_value,sizeof(float)*r1_dict_cnt,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy from p-value device to host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	
	err = hipFree(d_r1_dict_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	err = hipFree(d_p_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_p_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipFree(d_z_score);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_z_score (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	
	//Initialize the sortedIndex array, as sortedIndex will have the values sorted with quickSort based on descending order of Z-score
	//After sorting sortedIndex contains the new index of Z-score.
	int * sortedIndex = (int*) malloc(sizeof(int)*r1_dict_cnt);
	if(sortedIndex == NULL){
		printf("malloc error for sortedIndex\n");
	}
	for(i=0;i<r1_dict_cnt;i++){
		sortedIndex[i] = i;
	}

	quickSort(z_score_arr, sortedIndex, 0, r1_dict_cnt-1);
	
	printf("Write extracted compounds to output files\n");
	//Write to output files
	strcpy(filename1, argv[6]);
	strcat(filename1, "_temp_result1_Substance_compounds_cutoff_");
	strcat(filename1,cutoffstr);
	strcat(filename1,"_p_");
	strcat(filename1,pvaluestr);
	strcat(filename1,".txt");
	strcpy(filename2, argv[6]);
	strcat(filename2, "_temp_result1_Substance_compounds_cutoff_");
	strcat(filename2,cutoffstr);
	strcat(filename2,".txt");
	
	
	op1 = fopen(filename1, "w");
	fprintf(op1,"Term Pair\tMeSHID\tDistribution\tZ-Score\tP-value\n");
	op2 = fopen(filename2, "w");
	fprintf(op2,"Term Pair\tMeSHID\tDistribution\tZ-Score\tP-value\n");
	k=0;
	for(i=0;i<r1_dict_cnt;i++){
		if((r1_dict_value[(sortedIndex[i]*(sampleTimes+1))]>=cutoff) && (p_value_arr[sortedIndex[i]]<=p_value)){
			fprintf(op1,"%s;%s\t[",argv[6],r1_dict_keys[sortedIndex[i]].key);
			for(j=0;j<sampleTimes;j++)
				fprintf(op1,"%d, ",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j]);
			fprintf(op1,"%d]\t%f\t%f\n",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j], z_score_arr[i],p_value_arr[sortedIndex[i]]);
		}
		if((r1_dict_value[(sortedIndex[i]*(sampleTimes+1))]>=cutoff) && (p_value_arr[sortedIndex[i]]<=1.0)){
			fprintf(op2,"%s;%s\t[",argv[6],r1_dict_keys[sortedIndex[i]].key);
			for(j=0;j<=sampleTimes;j++)
				fprintf(op2,"%d, ",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j]);
			fprintf(op2,"%d]\t%f\t%f\n",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j], z_score_arr[i],p_value_arr[sortedIndex[i]]);
		}
		k++;
	}
	fclose(op1);
	fclose(op2);
	printf("Compounds output files written\n");

	free(r1_dict_keys);

	err = hipHostFree(p_value_arr);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host p_value_arr (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipHostFree(z_score_arr);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host z_score_arr (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	
	err = hipHostFree(r1_dict_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	free(sortedIndex);
	d_cinp1 = createDictionary();
	printf("Processing proteins\n");
	printf("Read input files\n");
	// Reading the dictionary of proteins of result 1 - 3rd argument
	inp_r1 = fopen(argv[3], rmode);
	if (inp_r1 == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[3]);
		exit(1);
	}	
	
	r1_dict_cnt = 0;

	while(1){
		fscanf(inp_r1, "%[^\n]%*c", str1);
		if( feof(inp_r1)) break;
		removeChar(str1,'\r');

		len = strlen(str1);
		for(i=0;(i<len);i++){
			char *newstr = (char*)malloc(len+1);
			if(newstr==NULL){
				printf("newstr malloc failed\n");
				exit(0);
			}
			j=0;
			while(str1[i] != '~'){
				newstr[j++] = str1[i++];
			}	
			newstr[j] = '\0';
			struct kvpair * e = searchDictionary(d_cinp1,newstr);
			if(e!=NULL){
				e->value++;
			}
			else{
				insertDictionary(d_cinp1,newstr,1);
				r1_dict_cnt++;
			}
			free(newstr);
		}
	}	
	fclose(inp_r1);

	// Reading the list of result 2- 4th argument
	inp_r2 = fopen(argv[4], rmode);
	if (inp_r2 == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[4]);
		exit(1);
	}	
	r2_cnt = 0;
	while (1) 
	{
		fscanf(inp_r2, "%[^\n]%*c", str1);
		if( feof(inp_r2)) break;
		removeChar(str1,'\r');
		inp2_list[r2_cnt] = (char*)malloc(strlen(str1)+1);
		if(inp2_list[r2_cnt]==NULL){
			printf("inp2_list[r2_cnt] malloc failed\n");
			exit(0);
		}
		strcpy(inp2_list[r2_cnt++],str1);		
	}
	fclose(inp_r2);
	printf("Input files read completed\n");
	//Sort inp2_list based on the number of tokens
	qsort(inp2_list,r2_cnt,sizeof(char *), sort);
	printf("Pre-process records for kernel launch\n");
	//pinned memory for optimized usage of memory transfer bandwidth
	err = hipHostMalloc((void**)&r1_dict_value, sizeof(int)*r1_dict_cnt*(sampleTimes+1), hipHostMallocDefault);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate r1_dict_value host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	//Process the input data for shipping
	free(r1_dict_keys);
	r1_dict_keys = (Node*)malloc(sizeof(Node)*r1_dict_cnt);
	if(r1_dict_keys==NULL){
		printf("r1_dict_keys malloc failed\n");
		exit(0);
	}
	j=0;
	k=0;
	for(i=0;i<d_cinp1->size;i++)
    {
    	if(d_cinp1->table[i]!=0){
			while(1)
			{
				strcpy(r1_dict_keys[j].key,d_cinp1->table[i]->key);
				r1_dict_keys[j].index = j;
				r1_dict_keys[j].next = NULL;
				r1_dict_value[j*(sampleTimes+1)] = d_cinp1->table[i]->value;
				j++;
				if(d_cinp1->table[i]->next!= NULL)
					d_cinp1->table[i] =  d_cinp1->table[i]->next;
				else
					break;
			}
		}
	}
	destroyDictionary(d_cinp1);
	//Strip off the number of tokens from list 2 records
	for(i=0;i<r2_cnt;i++){
		split0 = strtok_r(inp2_list[i], ";", &saveptr2);
	}

	temp1 = (char*) malloc(1000*r2_cnt*sizeof(char));
	if(temp1 == NULL){
		printf("temp1 malloc failed\n");
		exit(0);
	}
	j=0;
	for(i=0;i<r2_cnt;i++){
		for(k=0;k<1000;k++){
			if(k<strlen(inp2_list[i])){
				temp1[j++] = inp2_list[i][k];
			}
			else
				temp1[j++] = '^';
		}
		free(inp2_list[i]);
	}

	err = hipMalloc((void **)&d_r1_dict_keys,sizeof(Node)*r1_dict_cnt);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r1_dict_keys,r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Allocate global memory for list 2 records
	err = hipMalloc((void **)&d_r2_str,1000*sizeof(char)*r2_cnt);
	if(err != hipSuccess){
			fprintf(stderr,"Failed to allocate device d_r2_str (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r2_str,temp1,1000*sizeof(char)*r2_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r2_str (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Allocate global memory for index of dictionary 1 value list
	err = hipMalloc((void **)&d_r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r1_dict_value,r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}


	printf("Sampling for proteins begin\n");
	for(i=0;i<sampleTimes;i=i+256){
		threadCount = (r1_cnt>1024)?1024:r1_cnt;
		//sampleStrides: maximum number of strides every thread need to take for sampling
		sampleStrides = ceil(r1_cnt/threadCount);
		printf("Kernel deviceDDI launched with %d blocks of %d threads each\n", (sampleTimes-i)>256?256:(sampleTimes-i), threadCount);

		totalTime += elapsedTime;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord( start, 0 );
	
		deviceDDI<<<(sampleTimes-i)>256?256:(sampleTimes-i), threadCount, (sizeof(int)*(r1_dict_cnt))>>>( d_r2_str, d_r1_dict_value, sampleTimes, r1_cnt, r2_cnt, r1_dict_cnt, d_r1_dict_keys, d_hashtab, sampleStrides, threadCount,i,(i==0)?false:true);

		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &elapsedTime, start, stop );
		hipEventDestroy( start );
		hipEventDestroy( stop );
    	err = hipDeviceSynchronize();
		if(err != hipSuccess){
			fprintf(stderr,"Failed to launch deviceDDI kernel device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}
		err = hipGetLastError();
		if(err != hipSuccess){
			fprintf(stderr,"Failed to launch deviceDDI kernel device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}
		
		err = hipMemcpy(r1_dict_keys,d_r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyDeviceToHost);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_keys to host (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(d_r1_dict_keys,r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(r1_dict_value,d_r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyDeviceToHost);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_value from device to Host(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}

		err = hipMemcpy(d_r1_dict_value, r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_value from host to device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}

		err = hipMemcpy(hashtab,d_hashtab,sizeof(Node*)*5003,hipMemcpyDeviceToHost);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_hashtab to host (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(d_hashtab,hashtab,sizeof(Node*)*5003,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_hashtab (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(d_r2_str,temp1,1000*sizeof(char)*r2_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r2_str (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}
	}
	printf("Sampling completed\n");
    err = hipFree(d_r1_dict_keys);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
    err = hipFree(d_r2_str);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r2_str (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
	free(temp1);
    //Allocate Z-score array pinned memory for optimized usage of memory transfer bandwidth
    err = hipHostMalloc((void**)&z_score_arr, sizeof(float)*r1_dict_cnt, hipHostMallocDefault);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate z-score host (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

	//float * d_z_score_p;
    err = hipMalloc((void **)&d_z_score,sizeof(float)*r1_dict_cnt);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate device d_z_score (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    //Allocate P-value array pinned memory for optimized usage of memory transfer bandwidth
    err = hipHostMalloc((void**)&p_value_arr, sizeof(float)*r1_dict_cnt, hipHostMallocDefault);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate p-value host (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipMalloc((void **)&d_p_value,sizeof(float)*r1_dict_cnt);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate device d_p_value (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

	printf("Z-score, P-value calculation for proteins begin\n");
	printf("Kernel deviceZP launch with %d blocks of %d threads each\n", (int)ceil(r1_dict_cnt/256.0),256);
    totalTime += elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    deviceZP<<<ceil(r1_dict_cnt/256.0), 256>>>(d_r1_dict_value, sampleTimes, r1_dict_cnt, d_z_score, d_p_value);

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &elapsedTime, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	err = hipDeviceSynchronize();
	if(err != hipSuccess){
		fprintf(stderr,"Failed to launch deviceZP kernel device(error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE); 
	}
	err = hipGetLastError();
	if(err != hipSuccess){
		fprintf(stderr,"Failed to launch deviceZP kernel device(error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	printf("Z-score, P-value calculation for proteins completed\n");
	printf( "\n******** Total Running Time of Kernel = %0.5f seconds ******* \n", (elapsedTime+totalTime)/1000);
	printf("Copy output data to host memory\n");
	err = hipMemcpy(p_value_arr,d_p_value,sizeof(float)*r1_dict_cnt,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy from p-value device to host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	err = hipFree(d_r1_dict_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	err = hipFree(d_p_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_p_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(z_score_arr,d_z_score,sizeof(float)*r1_dict_cnt,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy from z-score device to host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	
	err = hipFree(d_z_score);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_z_score (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	//Initialize sortedIndex, this will hold correct index of the dictionary 1 records after sorting based on descending order of Z-score
	
	sortedIndex = (int*)malloc(sizeof(int)*r1_dict_cnt);
	if(sortedIndex == NULL){
		printf("sortedIndex malloc error\n");
	}
	for(i=0;i<r1_dict_cnt;i++){
		sortedIndex[i] = i;
	}
	//Sort the array based on descending order of Z-score
	quickSort(z_score_arr, sortedIndex, 0, r1_dict_cnt-1);
	//Write to output files
	strcpy(filename1, argv[6]);
	strcat(filename1, "_temp_result1_Substance_proteins_cutoff_");
	strcat(filename1,cutoffstr);
	strcat(filename1,"_p_");
	strcat(filename1,pvaluestr);
	strcat(filename1,".txt");
	strcpy(filename2, argv[6]);
	strcat(filename2, "_temp_result1_Substance_proteins_cutoff_");
	strcat(filename2,cutoffstr);
	strcat(filename2,".txt");

	printf("Write extracted proteins to output files\n");
	op1 = fopen(filename1, "w");
	fprintf(op1,"Term Pair\tMeSHID\tDistribution\tZ-Score\tP-value\n");
	op2 = fopen(filename2, "w");
	fprintf(op2,"Term Pair\tMeSHID\tDistribution\tZ-Score\tP-value\n");
	k=0;
	for(i=0;i<r1_dict_cnt;i++){
		
		if((r1_dict_value[(sortedIndex[i]*(sampleTimes+1))]>=cutoff) && (p_value_arr[sortedIndex[i]]<=p_value)){
			fprintf(op1,"%s;%s\t[",argv[6],r1_dict_keys[sortedIndex[i]].key);
			for(j=0;j<sampleTimes;j++)
				fprintf(op1,"%d, ",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j]);
			fprintf(op1,"%d]\t%f\t%f\n",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j], z_score_arr[i],p_value_arr[sortedIndex[i]]);
		}
		if((r1_dict_value[(sortedIndex[i]*(sampleTimes+1))]>=cutoff) && (p_value_arr[sortedIndex[i]]<=1.0)){
			fprintf(op2,"%s;%s\t[",argv[6],r1_dict_keys[sortedIndex[i]].key);
			for(j=0;j<=sampleTimes;j++)
				fprintf(op2,"%d, ",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j]);
			fprintf(op2,"%d]\t%f\t%f\n",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j], z_score_arr[i],p_value_arr[sortedIndex[i]]);
		}
		k++;
	}
	fclose(op1);
	fclose(op2);

	printf("Processing completed\n");
	free(r1_dict_keys);
	err = hipHostFree(p_value_arr);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host p_value_arr (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipHostFree(z_score_arr);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host z_score_arr (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipHostFree(r1_dict_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	free(sortedIndex);
	free(hashtab);

	err = hipFree(d_hashtab);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_hashtab (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
}
//End of program