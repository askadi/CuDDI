
/* Execution Format : ./<exe> <drug_result_1_dict_compounds.txt> <drug_result_2_dict_compounds.txt> <drug_result_1_dict_proteins.txt> <drug_result_2_dict_proteins.txt> <para.txt> <drug name>
*/

#include <stdio.h>
#include <errno.h>
#include <math.h>
#include <string.h>
#include <unistd.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/dir.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define INITIAL_SIZE (102397)
#define MULTIPLIER (97)

typedef struct dict *Dict;

/* create a new empty dictionary */
Dict DictCreate(void);
/* destroy a dictionary */
void DictDestroy(Dict);
/* insert a new key-value pair into an existing dictionary */
void DictInsert(Dict, const char *key, int val);
/* return the most recently inserted value associated with a key */
/* or 0 if no matching key is present */
struct elt * DictSearch(Dict, const char *key);
/* delete the most recently inserted record with the given key */
/* if there is no such record, has no effect */
void DictDelete(Dict, const char *key);

// Dictionary Code from : http://www.cs.yale.edu/homes/aspnes/pinewiki/C(2f)HashTables.html?highlight=(CategoryAlgorithmNotes)

//Structure for nodes in dictionary
struct elt {
    struct elt *next;
    char *key;
    int value;
};

//Structure for dictionary
struct dict {
    int size;           /* size of the pointer table */
    int n;              /* number of elements stored */
    struct elt **table;
};

//Structure for nodes in the CUDA hashtable
typedef struct node {
	char key[80];
	int index;
	struct node *next;
} Node;

/* dictionary initialization code used in both DictCreate and grow */
Dict internalDictCreate(int size)
{
    Dict d;
    int i;
    d = (Dict)malloc(sizeof(*d));
	if(d==NULL){
		printf("d malloc failed\n");
		exit(0);
	}
    assert(d != 0);
    d->size = size;
    d->n = 0;
    d->table = (elt **)malloc(sizeof(struct elt *) * d->size);
	if(d->table==NULL){
		printf("d->table malloc failed\n");
		exit(0);
	}
    assert(d->table != 0);
    for(i = 0; i < d->size; i++) 
    	d->table[i] = 0;
    return d;
}

//Function to create dictionary
Dict DictCreate()
{
    return internalDictCreate(INITIAL_SIZE);
}

//Function to free dictionary
void DictDestroy(Dict d)
{
    int i;
    struct elt *e;
    struct elt *next;
    for(i = 0; i < d->size; i++) {
        for(e = d->table[i]; e != 0; e = next) {
            next = e->next;
            free(e->key);
            //free(e->value);
            free(e);
        }
    }
    free(d->table);
    free(d);
}

//Function to compute hash value
static unsigned long hash_function(const char *s)
{
    unsigned const char *us;
    unsigned long h;
    h = 0;
    for(us = (unsigned const char *) s; *us; us++) {
        h = h * MULTIPLIER + *us;
    }
    return h;
}
/* insert a new key-value pair into an existing dictionary */
void DictInsert(Dict d, const char *key, int val)
{
    struct elt *e;
    unsigned long h;
    assert(key);
    assert(val);
    e = (elt*)malloc(sizeof(*e));
	if(e==NULL){
		printf("e malloc failed\n");
		exit(0);
	}
    assert(e);
    e->key = strdup(key);
    //e->value = (int*)malloc(sizeof(int));
    e->value = val;
    h = hash_function(key) % d->size;
    e->next = d->table[h];
    d->table[h] = e;
    d->n++;
	return;
}

/* return the most recently inserted Node associated with a key */
/* or NULL if no matching key is present */
struct elt * DictSearch(Dict d, const char *key)
{
    struct elt *e;
    for(e = d->table[hash_function(key) % d->size]; e != 0; e = e->next) {
        if(!strcmp(e->key, key)) {
            /* got it */
            return e;
        }
    }
    return NULL;
}

/* delete the most recently inserted record with the given key */
/* if there is no such record, has no effect */
void DictDelete(Dict d, const char *key)
{
    struct elt **prev;          /* what to change when elt is deleted */
    struct elt *e;              /* what to delete */
    for(prev = &(d->table[hash_function(key) % d->size]); 
        *prev != 0; 
        prev = &((*prev)->next)) {
        if(!strcmp((*prev)->key, key)) {
            /* got it */
            e = *prev;
            *prev = e->next;
            free(e->key);
            //free(e->value);
            free(e);
            return;
        }
    }
}

//Function to compute normal distribution of a value, equivalent to Python's CDF.norm from NVIDIA CUDA samples
//http://stackoverflow.com/questions/2328258/cumulative-normal-distribution-function-in-c-c

__device__ float CND(float d)
{
    const double       A1 = 0.31938153;
    const double       A2 = -0.356563782;
    const double       A3 = 1.781477937;
    const double       A4 = -1.821255978;
    const double       A5 = 1.330274429;
    const double RSQRT2PI = 0.39894228040143267793994605993438;
    double
    K = 1.0 / (1.0 + 0.2316419 * fabs(d));
    double
    cnd = RSQRT2PI * exp(- 0.5 * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));
    if (d > 0)
        cnd = 1.0 - cnd;
    return cnd;
}

//Function to remove specific characters from input string, used to remove carriage returns
void removeChar(char *str, char garbage) {
    char *src, *dst;
    for (src = dst = str; *src != '\0'; src++) {
        *dst = *src;
        if (*dst != garbage) dst++;
    }
    *dst = '\0';
	return;
}

//Function used by qsort to sort the records based on number of tokens
int sort(const void* a, const void* b)
{

	 char *ia = strdup(*(const char **)a);
     char *ib = strdup(*(const char **)b);
	 char *split1, *saveptr, *saveptr1;
	 split1 = strtok_r(ia, ";", &saveptr);
	 split1 = strtok_r(NULL, ";", &saveptr);
	 int x = atoi(split1);
	 split1 = strtok_r(ib, ";", &saveptr1);
	 split1 = strtok_r(NULL, ";", &saveptr1);
	 return (x-atoi(split1));
}

/*Kernel function performs sampling and Z-score, P-value calculation
It is designed in this way: One block does one sampling and every thread processes one record. In case number of records to be processed exceed 1024, then some threads will take more than one stride. That is, some threads process more than one record during one sampling.
Stages in kernel function are: 
1) Build the dictionary 'd_hashtab', for O(1) time lookup of keyword while sampling, first thread will ensure all the keys 'd_r1_dict_keys' are linked in the hashtable.
2) Generate the random numbers and sort them. Every thread will generate a random number and first thread in every block will sort the random numbers using iterative quick sort function. In case number of random numbers required are higher than 1024, then some threads will take more strides to generate the required number of random numbers. 
3) Shared memory initialization for sampling. This is required, as during sampling if keyword is found then we increment the count.
4) Sampling, every block performs one sampling. And, every thread will process atleast one record. That is, thread will extract the keywords/tokens in the record and then finds for the keyword in the dictionary, if found then shared memory is incremented.
5) Copy data to global memory from shared memory for Z-score and P-value calculation.
6) Z-score and P-value calculation. In this stage, one thread will compute Z-score and P-value for atleast one record. In case, the number of threads are fewer than number of records, then some threads will take more strides to compute the Z-score and P-value.
Generally, this would never happen, because the number of threads (blocks*threadCount) is always higher than the number of records.
Threads operate on the 'd_r1_dict_value', producing Z-score and P-value for each array of integers. 

Note: Shared memory s_random_r1_key array has 2 types of value: random numbers and array for keywords found during sampling.
	  First part is the random numbers, second part is used for incrementing array of keywords for sampling.

Arguments passed to kernel function:
	  * d_r1_dict_list - is the list of keywords from dictionary 1, created in the CPU.
	  * d_r2_str - is the list of records to be used for sampling.
	  * d_r1_dict_value - Global values for vector produced from sampling.
	  * sampleTimes - number of samples.
	  * sampleSize - size of the sample.
	  * randomRange - maximum value of each random number.
	  * r1_dict_cnt - number of keywords in dictionary 1.
	  * d_z_score - array to hold Z-scores.
	  * d_p_value - array to hold P-values.
	  * d_r1_dict_keys - keywords of dictionary 1, to populate hashtable in kernel function.
	  * sampleStrides - maximum number of strides every thread will take for sampling.
	  * threadCount - number of threads per block.
*/
__global__ void deviceDDI(char * d_r2_str, int * d_r1_dict_value, int sampleTimes, int sampleSize, int randomRange, int r1_dict_cnt, Node *d_r1_dict_keys, int sampleStrides, int threadCount, int samplesCompleted){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j, k, x = 0, ind1, random;
	unsigned hashval;

	extern __shared__ int s_r1_value_list[];
    __shared__ Node *s_hashtab[5003];

	//s_random_r1_key array has 2 types of value: random numbers and array for keywords found during sampling.
	//First part is the random numbers, second part is used for incrementing array of keywords for sampling.
	//Build the dictionary
	if(threadIdx.x==0){
		//Initialize the hash table
		for(j=0;j<5003;j++){
            s_hashtab[j] = 0;
		}
		j=0;
		k=0;
		//Build the dictionary
		/*while(d_r1_dict_list[k]!='^'){
			x=0;
			hashval = 0;
			while(d_r1_dict_list[k]!='~'){
				d_r1_dict_keys[j].key[x++] = d_r1_dict_list[k];
				hashval = ((int)d_r1_dict_list[k++] + 97*hashval)%5003;
			}
			d_r1_dict_keys[j].key[x] = '\0';
			d_r1_dict_keys[j].index = j;
			d_r1_dict_keys[j].next = (s_hashtab[hashval]==0?NULL:s_hashtab[hashval]);
			s_hashtab[hashval] = &d_r1_dict_keys[j++];
			k++;
		}*/
		for(j=0;j<r1_dict_cnt;j++){
			x=0;
			hashval = 0;
			while(d_r1_dict_keys[j].key[x]!='\0'){
				hashval = ((int)d_r1_dict_keys[j].key[x++] + 97*hashval)%5003;
			}
			d_r1_dict_keys[j].next = (s_hashtab[hashval]==0?NULL:s_hashtab[hashval]);
			s_hashtab[hashval] = &d_r1_dict_keys[j];
		}
		//Initialize the shared memory
		for(j=0;j<(r1_dict_cnt);j++){
 			s_r1_value_list[j] = 0;
 		}
	}	
	__syncthreads();		
	//Generate the random numbers
	hiprandState_t state;
 	hiprand_init(clock64(), i, 0, &state);		
 	//Sampling
 	for(j=0;(j<sampleStrides)&&((threadIdx.x+(j*threadCount))<sampleSize);j++){
 		char str_split[80];
 		x=0;
 		hashval = 0;
		random = hiprand(&state)%randomRange;
		for(k=random*1000;k<random*1000+1000;k++){
			if(d_r2_str[k] == '^')
				break;
 			if(d_r2_str[k] != '~'){
 		 		str_split[x++] = d_r2_str[k];
 		 		hashval = ((int)d_r2_str[k] + 97*hashval)%5003;
 			}
 			else{
 				str_split[x] = '\0';
				Node *np = s_hashtab[hashval];
 				while((np!= NULL)&&(np!=0)){
 		 			ind1 = 0;
 		 			while((np->key[ind1] != '\0')&&(ind1<x)){
 		 				if(np->key[ind1] == str_split[ind1])
 		 					ind1++;
 		 				else
 		 					break;
 		 			}
 		 			if((np->key[ind1] == '\0')&&(ind1==x)){
						atomicAdd(&s_r1_value_list[(np->index)],1);
 		 				break;
 		 			}
 		 			if(np->next == NULL||np->next==0)
 		 				break;
 		 			np = np->next;
 				}
 				x=0;
 				hashval = 0;
 			}
 		}
 	}
	__syncthreads(); 

	//Copy to global memory from shared memory
	if(threadIdx.x==0){
		for(j=0;j<(r1_dict_cnt);j++){
 			d_r1_dict_value[(j)*(sampleTimes+1)+blockIdx.x+1+samplesCompleted] = s_r1_value_list[j];
 		}
	}
}
__global__ void deviceZP(int * d_r1_dict_value, int sampleTimes,int r1_dict_cnt, float * d_z_score, float * d_p_value){
    int x, j,  i = blockDim.x * blockIdx.x + threadIdx.x;
    float mean =0,sd;
    if(i<r1_dict_cnt){
            x = 0;
            sd = 0;
            for(j=1;j<=sampleTimes;j++){
                x += d_r1_dict_value[(i*(sampleTimes+1))+j];
            }
            mean = x/(sampleTimes);
            for(j=1;j<=sampleTimes;j++){
                sd += (d_r1_dict_value[(i*(sampleTimes+1))+j]-mean)*(d_r1_dict_value[(i*(sampleTimes+1))+j]-mean);
            }
            sd = sqrt(sd/(sampleTimes));
            //if(std != 0)
            //Better approach to check if standard deviation is equal to zero or not, because standard deviation is floating point
            if(fabs(sd)>pow(10.0,-7))
                d_z_score[i] = (d_r1_dict_value[i*(sampleTimes+1)] - mean)/sd;
            else{
                if(d_r1_dict_value[i*(sampleTimes+1)] != (int)mean)
                    d_z_score[i] = d_r1_dict_value[i*(sampleTimes+1)]*100;
                else
                    d_z_score[i] = -100;
            }
            d_p_value[i] = 1-CND(d_z_score[i]);
			//printf("Z-score = %f, p-value = %f\n",d_z_score[i],d_p_value[i]);
    }
}
//Function to partition records while sorting based on Z-score, called by quickSort
int partition( float a[], int index[], int l, int r) {
   int i, j, t;
   float temp;
   float pivot = a[l];
   i = l; j = r+1;		
   while( 1)
   {
   	do ++i; while( a[i] >= pivot && i <= r );
   	do --j; while( a[j] < pivot );
   	if( i >= j ) break;
   	temp = a[i]; a[i] = a[j]; a[j] = temp;
   	t = index[i];
   index[i] = index[j];
   index[j] = t;
   }
   temp = a[l]; a[l] = a[j]; a[j] = temp;
   t = index[l];
   index[l] = index[j];
   index[j] = t;
   return j;
}

//Function to quicksort the records based on Z-score
void quickSort(float a[], int index[], int l, int r)
{
   int j;
   if( l < r ) 
   {
       j = partition( a, index, l, r);
       quickSort( a, index, l, j-1);
       quickSort( a, index, j+1, r);
   }	
   return;
}


int main(int argc, char *argv[])
{
	if(argc!=7){
		printf("\nIncorrect arguments passed, Please pass <Compounds with interactions>, <Compounds without interactions>, <Proteins with interactions>, <Proteins without interactions>, <PMID Substances>, <para.txt>, <Drug Name> as arguments\n");
		exit(1);
	}
	FILE *inp_r1, *inp_r2, *inp_para, *op1, *op2;
	char rmode[2] = "r";
	char str1[10000];
	char *split0,*split1, *saveptr, *saveptr1, *saveptr2;
	char *inp2_list[100000];
	//char * r1_dict_list;
	char filename1[100], filename2[100], cutoffstr[20], pvaluestr[20];
	size_t len = 0;
	Dict d_cinp1;
	int cutoff, sampleTimes;
	float p_value, elapsedTime, totalTime=0;
	//float z_score;
	int i=0, j=0, k=0, x;
	int r1_cnt, r2_cnt, r1_dict_cnt, threadCount, sampleStrides;
	printf("Drug name = %s\n",argv[6]);
	printf("Read input files\n");
	hipEvent_t start, stop;
	d_cinp1 = DictCreate();			

	//Read the parameters from para.txt - 4th argument
	inp_para = fopen(argv[5],rmode);
	if (inp_para == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[5]);
		exit(1);
	}	
	while(1)
	{	
		fscanf(inp_para,"%[^\n]%*c", str1);
		if(feof(inp_para)) break;
		split0 = strtok_r(str1, "\t", &saveptr);
		split1 = strtok_r(NULL, "\t", &saveptr);
	 	removeChar(split0,'\r');
		removeChar(split1,'\r');
		if( strcmp(split0,"sampleTimes") == 0)
		{
			char temp[20];
			strcpy(temp, split1);
			sampleTimes = atoi(temp);	
		}
		
		 else if( strcmp(split0,"cutoff") == 0)
		{
			char temp[20];
			strcpy(temp, split1);
			strcpy(cutoffstr,temp);
			cutoff = atoi(temp);	
		}
		else if( strcmp(split0,"p_value") == 0)
		{
			char temp[20];
			strcpy(temp, split1);
			strcpy(pvaluestr,temp);
			p_value = atof(temp);	
		}
		/*else if( strcmp(split0,"z_score") == 0)
		{
			char temp[20];
			strcpy(temp, split1);
			z_score = atof(temp);	
		}*/
	}
	fclose(inp_para);
	printf("Number of Samples = %d\n",sampleTimes);
	if(sampleTimes <=0){
		printf("Incorrect number of samples specified = %d, value of atleast 1 is expected\n", sampleTimes);
		exit(0);
	}
	// Reading the dictionary of compounds of result 1 - 1st argument
	// Create and populate dictionary 'd_cinp1' while reading the records
	inp_r1 = fopen(argv[1], rmode);
	if (inp_r1 == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[1]);
		exit(1);
	}	
	
	r1_cnt = 0;
	r1_dict_cnt = 0;

	while(1){
		fscanf(inp_r1, "%[^\n]%*c", str1);
		if( feof(inp_r1)) break;
		removeChar(str1,'\r');
		r1_cnt++;
		len = strlen(str1);
		for(i=0;(i<len);i++){
			char *newstr = (char*)malloc(len+1);
			if(newstr==NULL){
				printf("malloc to newstr failed\n");
				exit(0);
			}
			j=0;
			while(str1[i] != '~'){
				newstr[j++] = str1[i++];
			}	
			newstr[j] = '\0';
			struct elt * e = DictSearch(d_cinp1,newstr);
			if(e!=NULL){
				e->value++;
			}
			else{
				DictInsert(d_cinp1,newstr,1);
				r1_dict_cnt++;
			}
			free(newstr);
		}
	}	
	fclose(inp_r1);

	// Reading the list of result 2- 2nd argument
	inp_r2 = fopen(argv[2], rmode);
	if (inp_r2 == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[2]);
		exit(1);
	}	
	r2_cnt = 0;
	while (1) 
	{
		fscanf(inp_r2, "%[^\n]%*c", str1);
		if( feof(inp_r2)) break;
		removeChar(str1,'\r');
		inp2_list[r2_cnt] = (char*)malloc(strlen(str1)+1);
		if(inp2_list[r2_cnt]==NULL){
			printf("malloc to inp2_list[r2_cnt] failed\n");
			exit(0);
		}
		strcpy(inp2_list[r2_cnt++],str1);		
	}
	fclose(inp_r2);
	printf("Input files read completed\n");
	printf("Sample size = %d\n", r1_cnt);
	//Sort inp2_list based on the number of tokens or length
	qsort(inp2_list,r2_cnt,sizeof(char *), sort);

	printf("Pre-process records for kernel launch\n");
	//r1_dict_list = (char*)malloc(80*r1_dict_cnt);

	//populate value list for dictionary 1
	hipError_t err = hipSuccess;
	int * r1_dict_value;
	//pinned memory for optimized usage of memory transfer bandwidth
	err = hipHostMalloc((void**)&r1_dict_value, sizeof(int)*r1_dict_cnt*(sampleTimes+1), hipHostMallocDefault);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate r1_dict_value host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Device value list for dictionary 1
	int *d_r1_dict_value;
	Node * r1_dict_keys = (Node*)malloc(sizeof(Node)*r1_dict_cnt);
	if(r1_dict_keys==NULL){
		printf("malloc to r1_dict_keys failed\n");
		exit(0);
	}
	j=0;
	k=0;
	for(i=0;i<d_cinp1->size;i++)
    {
    	if(d_cinp1->table[i]!=0){
			while(1)
			{
				//char * key = strdup(d_cinp1->table[i]->key);
				//int subind = 0;
				/*for(subind=0;subind<strlen(key);subind++){
					r1_dict_list[k++] = key[subind];
				}
				r1_dict_list[k++] = '~';
				r1_dict_value[j*(sampleTimes+1)] = d_cinp1->table[i]->value;
				*/
				//for(subind=0;subind<strlen(key);subind++){
					//r1_dict_keys[j].key[x++] = key[subind];
				//}
				strcpy(r1_dict_keys[j].key,d_cinp1->table[i]->key);
				r1_dict_keys[j].index = j;
				r1_dict_keys[j].next = NULL;
				r1_dict_value[j*(sampleTimes+1)] = d_cinp1->table[i]->value;
				j++;
				//free(key);
				if(d_cinp1->table[i]->next!= NULL)
					d_cinp1->table[i] =  d_cinp1->table[i]->next;
				else
					break;
			}
		}
	}
	//r1_dict_list[k] = '^';
	DictDestroy(d_cinp1);
	/*x=0;
	while(r1_dict_list[x] != '^'){
		while(r1_dict_list[x] != '~')
			printf("%c",r1_dict_list[x++]);
		printf("\n");
		x++;
	}*/
	//Strip off the number of tokens from every record in list 2.
	for(i=0;i<r2_cnt;i++){
		split0 = strtok_r(inp2_list[i], ";", &saveptr1);
	}

	//Process the records for shipping to kernel
	char * temp1 = (char*) malloc(1000*r2_cnt*sizeof(char));
	if(temp1==NULL){
		printf("temp1 malloc failed\n");
		exit(0);
	}
	char * d_r2_str;
	j=0;
	for(i=0;i<r2_cnt;i++){
		for(k=0,x=0;k<1000;k++){
			while(x<strlen(inp2_list[i])){
				temp1[j++] = inp2_list[i][x++];
				k++;
			}
			temp1[j++] = '^';
		}
		free(inp2_list[i]);
	}

	//char* d_r1_dict_list;
	//Allocate global memory for dictionary 1 keywords
	/*err = cudaMalloc((void **)&d_r1_dict_list,80*sizeof(char)*r1_dict_cnt);
	if(err != cudaSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_list (error code %s) !\n", cudaGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = cudaMemcpy(d_r1_dict_list,r1_dict_list,80*sizeof(char)*r1_dict_cnt,cudaMemcpyHostToDevice);
	if(err != cudaSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_list (error code %s) !\n", cudaGetErrorString(err));
		exit(EXIT_FAILURE);	
	}*/
	
	//Allocate global memory for input list 2 records
	err = hipMalloc((void **)&d_r2_str,1000*sizeof(char)*r2_cnt);
	if(err != hipSuccess){
			fprintf(stderr,"Failed to allocate device d_r2_str (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r2_str,temp1,1000*sizeof(char)*r2_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r2_str (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Allocate global memory for dictionary 1 value list 
	err = hipMalloc((void **)&d_r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r1_dict_value,r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	Node * d_r1_dict_keys = NULL;
	err = hipMalloc((void **)&d_r1_dict_keys,sizeof(Node)*r1_dict_cnt);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	err = hipMemcpy(d_r1_dict_keys,r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	printf("Sampling for compounds begin\n");
	for(i=0;i<sampleTimes;i=i+256){

		threadCount = (r1_cnt>1024)?1024:r1_cnt;
		//sampleStrides: maximum number of strides every thread needs to take for sampling
		sampleStrides = ceil(r1_cnt/threadCount);
		printf("Kernel deviceDDI launched with %d blocks of %d threads each\n", (sampleTimes-i)>256?256:(sampleTimes-i), threadCount);

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord( start, 0 );

		deviceDDI<<<(sampleTimes-i)>256?256:(sampleTimes-i), threadCount, (sizeof(int)*(r1_dict_cnt))>>>(d_r2_str, d_r1_dict_value, sampleTimes, r1_cnt, r2_cnt, r1_dict_cnt, d_r1_dict_keys, sampleStrides, threadCount,i);
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &elapsedTime, start, stop );
		hipEventDestroy( start );
		hipEventDestroy( stop );

		err = hipGetLastError();
		if(err != hipSuccess){
			fprintf(stderr,"Failed to launch deviceDDI kernel device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}

		err = hipMemcpy(r1_dict_value,d_r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyDeviceToHost);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_value from device to Host(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}

		err = hipMemcpy(d_r1_dict_value, r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_value from host to device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}
		
		err = hipMemcpy(d_r1_dict_keys,r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(d_r2_str,temp1,1000*sizeof(char)*r2_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r2_str (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

	}
	printf("Sampling for compounds completed\n");

    err = hipFree(d_r2_str);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r2_str (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipFree(d_r1_dict_keys);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
    free(temp1);

    float *d_z_score;
    float *z_score_arr;
    //Allocate array for Z-score, pinned memory for optimized usage of memory transfer bandwidth
    err = hipHostMalloc((void**)&z_score_arr, sizeof(float)*r1_dict_cnt, hipHostMallocDefault);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate z-score host (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipMalloc((void **)&d_z_score,sizeof(float)*r1_dict_cnt);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate device d_z_score (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    float *d_p_value;
    float *p_value_arr;
    //Allocate array for P-value, pinned memory for optimized usage of memory transfer bandwidth
    err = hipHostMalloc((void**)&p_value_arr, sizeof(float)*r1_dict_cnt, hipHostMallocDefault);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate p-value host (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipMalloc((void **)&d_p_value,sizeof(float)*r1_dict_cnt);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate device d_p_value (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
	
	printf("Z-score and P-value calculation for Compounds begin\n");
	printf("Kernel deviceZP launched with %d blocks of %d threads each\n", (int)ceil(r1_dict_cnt/256.0), 256);
    totalTime += elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    deviceZP<<<ceil(r1_dict_cnt/256.0), 256>>>(d_r1_dict_value, sampleTimes, r1_dict_cnt, d_z_score, d_p_value);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &elapsedTime, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	err = hipGetLastError();
	if(err != hipSuccess){
		fprintf(stderr,"Failed to launch deviceZP kernel device(error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	printf("Z-score, P-value calculation completed\n");
	err = hipMemcpy(z_score_arr,d_z_score,sizeof(float)*r1_dict_cnt,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy from z-score device to host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(p_value_arr,d_p_value,sizeof(float)*r1_dict_cnt,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy from p-value device to host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	
	err = hipFree(d_r1_dict_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	err = hipFree(d_p_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_p_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipFree(d_z_score);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_z_score (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	
	//Initialize the sortedIndex array, as sortedIndex will have the values sorted with quickSort based on descending order of Z-score
	//After sorting sortedIndex contains the new index of Z-score.
	int sortedIndex[r1_dict_cnt];
	for(i=0;i<r1_dict_cnt;i++){
		sortedIndex[i] = i;
	}

	quickSort(z_score_arr, sortedIndex, 0, r1_dict_cnt-1);
	//Read PMID list file and populate dictionary d_pmid
	/*inp_pmid = fopen(argv[5], rmode);
	if (inp_pmid == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[5]);
		exit(1);
	}	
	while (1) 
	{
		fscanf(inp_pmid, "%[^\n]%*c", str1);
		removeChar(str1,'\r');
		if(feof(inp_pmid)) break;
		split0 = strtok_r(str1, ";", &saveptr);
		split1 = strtok_r(NULL, ";", &saveptr);
		DictInsertC(d_drugs_app_wdrwn, split0, split1);		
	}	
	fclose(inp_pmid);
	*/
	printf("Write extracted compounds to output files\n");
	//Write to output files
	strcpy(filename1, argv[6]);
	strcat(filename1, "_temp_result1_Substance_compounds_cutoff_");
	strcat(filename1,cutoffstr);
	strcat(filename1,"_p_");
	strcat(filename1,pvaluestr);
	strcat(filename1,".txt");
	strcpy(filename2, argv[6]);
	strcat(filename2, "_temp_result1_Substance_compounds_cutoff_");
	strcat(filename2,cutoffstr);
	strcat(filename2,".txt");
	
	
	op1 = fopen(filename1, "w");
	fprintf(op1,"Term Pair\tMeSHID\tDistribution\tZ-Score\tP-value\n");
	op2 = fopen(filename2, "w");
	fprintf(op2,"Term Pair\tMeSHID\tDistribution\tZ-Score\tP-value\n");
	k=0;
	for(i=0;i<r1_dict_cnt;i++){
		//char key[80];
		//j=0;
		/*while(r1_dict_list[k] != '~'){
			key[j++] = r1_dict_list[k++];
		}
		key[j] = '\0';
		*/
		if((r1_dict_value[(sortedIndex[i]*(sampleTimes+1))]>=cutoff) && (p_value_arr[sortedIndex[i]]<=p_value)){
			fprintf(op1,"%s;%s\t[",argv[6],r1_dict_keys[i].key);
			for(j=0;j<sampleTimes;j++)
				fprintf(op1,"%d, ",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j]);
			fprintf(op1,"%d]\t%f\t%f\n",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j], z_score_arr[i],p_value_arr[sortedIndex[i]]);
		}
		if((r1_dict_value[(sortedIndex[i]*(sampleTimes+1))]>=cutoff) && (p_value_arr[sortedIndex[i]]<=1.0)){
			fprintf(op2,"%s;%s\t[",argv[6],r1_dict_keys[i].key);
			for(j=0;j<=sampleTimes;j++)
				fprintf(op2,"%d, ",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j]);
			fprintf(op2,"%d]\t%f\t%f\n",r1_dict_value[(sortedIndex[i]*(sampleTimes+1))+j], z_score_arr[i],p_value_arr[sortedIndex[i]]);
		}
		k++;
	}
	fclose(op1);
	fclose(op2);
	printf("Compounds output files written\n");

	//free(r1_dict_list);
	free(r1_dict_keys);

	err = hipHostFree(p_value_arr);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host p_value_arr (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipHostFree(z_score_arr);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host z_score_arr (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	
	err = hipHostFree(r1_dict_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	d_cinp1 = DictCreate();
	printf("Processing proteins\n");
	printf("Read input files\n");
	// Reading the dictionary of proteins of result 1 - 3rd argument
	inp_r1 = fopen(argv[3], rmode);
	if (inp_r1 == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[3]);
		exit(1);
	}	
	
	//r1_cnt = 0;
	r1_dict_cnt = 0;

	while(1){
		fscanf(inp_r1, "%[^\n]%*c", str1);
		if( feof(inp_r1)) break;
		removeChar(str1,'\r');
		//r1_cnt++;
		len = strlen(str1);
		for(i=0;(i<len);i++){
			char *newstr = (char*)malloc(len+1);
			if(newstr==NULL){
				printf("newstr malloc failed\n");
				exit(0);
			}
			j=0;
			while(str1[i] != '~'){
				newstr[j++] = str1[i++];
			}	
			newstr[j] = '\0';
			struct elt * e = DictSearch(d_cinp1,newstr);
			if(e!=NULL){
				e->value++;
			}
			else{
				DictInsert(d_cinp1,newstr,1);
				r1_dict_cnt++;
			}
			free(newstr);
		}
	}	
	fclose(inp_r1);

	// Reading the list of result 2- 4th argument
	inp_r2 = fopen(argv[4], rmode);
	if (inp_r2 == NULL) 
	{
		fprintf(stderr, "Can't open input file %s!\n", argv[4]);
		exit(1);
	}	
	r2_cnt = 0;
	while (1) 
	{
		fscanf(inp_r2, "%[^\n]%*c", str1);
		if( feof(inp_r2)) break;
		removeChar(str1,'\r');
		inp2_list[r2_cnt] = (char*)malloc(strlen(str1)+1);
		if(inp2_list[r2_cnt]==NULL){
			printf("inp2_list[r2_cnt] malloc failed\n");
			exit(0);
		}
		strcpy(inp2_list[r2_cnt++],str1);		
	}
	fclose(inp_r2);
	printf("Input files read completed\n");
	//Sort inp2_list based on the number of tokens
	qsort(inp2_list,r2_cnt,sizeof(char *), sort);

	printf("Pre-process records for kernel launch\n");
	//r1_dict_list = (char*)malloc(80*r1_dict_cnt);

	//populate value list for dictionary 1
	//int r1_dict_value[r1_dict_cnt*(sampleTimes+1)];

	//pinned memory for optimized usage of memory transfer bandwidth
	err = hipHostMalloc((void**)&r1_dict_value, sizeof(int)*r1_dict_cnt*(sampleTimes+1), hipHostMallocDefault);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate r1_dict_value host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Process the input data for shipping
	free(r1_dict_keys);
	r1_dict_keys = (Node*)malloc(sizeof(Node)*r1_dict_cnt);
	if(r1_dict_keys==NULL){
		printf("r1_dict_keys malloc failed\n");
		exit(0);
	}
	j=0;
	k=0;
	for(i=0;i<d_cinp1->size;i++)
    {
    	if(d_cinp1->table[i]!=0){
			while(1)
			{
				//char * key = strdup(d_cinp1->table[i]->key);
				/*ind2_r1_dict_key[2*j] = k;
				k += strlen(key);
				ind2_r1_dict_key[(2*j)+1] = k;
				*/
				//int subind = 0;
				//for(x=ind2_r1_dict_key[2*j];x<ind2_r1_dict_key[(2*j)+1];x++){
				/*for(subind=0;subind<strlen(key);subind++){
					r1_dict_list[k++] = key[subind];
				}
				r1_dict_list[k++] = '~';
				r1_dict_value[j*(sampleTimes+1)] = d_cinp1->table[i]->value;
				*/
				strcpy(r1_dict_keys[j].key,d_cinp1->table[i]->key);
				r1_dict_keys[j].index = j;
				r1_dict_keys[j].next = NULL;
				r1_dict_value[j*(sampleTimes+1)] = d_cinp1->table[i]->value;

				j++;
				//free(key);
				if(d_cinp1->table[i]->next!= NULL)
					d_cinp1->table[i] =  d_cinp1->table[i]->next;
				else
					break;
			}
		}
	}
	//r1_dict_list[k] = '^';
	DictDestroy(d_cinp1);
	//Strip off the number of tokens from list 2 records
	for(i=0;i<r2_cnt;i++){
		split0 = strtok_r(inp2_list[i], ";", &saveptr2);
	}

	temp1 = (char*) malloc(1000*r2_cnt*sizeof(char));
	if(temp1 == NULL){
		printf("temp1 malloc failed\n");
		exit(0);
	}
	j=0;
	for(i=0;i<r2_cnt;i++){
		/*ind2_r2_str[2*i] = k;
		k += strlen(inp2_list[i]);
		ind2_r2_str[(2*i)+1] = k;
		*/
		//for(j=ind2_r2_str[2*i];j<ind2_r2_str[(2*i)+1];j++){
		for(k=0,x=0;k<1000;k++){
			while(x<strlen(inp2_list[i])){
				temp1[j++] = inp2_list[i][x++];
				k++;
			}
			if(k<1000)
				temp1[j++] = '^';
		}
		free(inp2_list[i]);
	}

	err = hipMalloc((void **)&d_r1_dict_keys,sizeof(Node)*r1_dict_cnt);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r1_dict_keys,r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Allocate global memory for list 2 records
	err = hipMalloc((void **)&d_r2_str,1000*sizeof(char)*r2_cnt);
	if(err != hipSuccess){
			fprintf(stderr,"Failed to allocate device d_r2_str (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r2_str,temp1,1000*sizeof(char)*r2_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r2_str (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//Allocate global memory for index of dictionary 1 value list
	err = hipMalloc((void **)&d_r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to allocate device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipMemcpy(d_r1_dict_value,r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}


	printf("Sampling for proteins begin\n");
	for(i=0;i<sampleTimes;i=i+256){
		threadCount = (r1_cnt>1024)?1024:r1_cnt;
		//sampleStrides: maximum number of strides every thread need to take for sampling
		sampleStrides = ceil(r1_cnt/threadCount);
		printf("Kernel deviceDDI launched with %d blocks of %d threads each\n", (sampleTimes-i)>256?256:(sampleTimes-i), threadCount);

		totalTime += elapsedTime;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord( start, 0 );
	
		deviceDDI<<<(sampleTimes-i)>256?256:(sampleTimes-i), threadCount, (sizeof(int)*(r1_dict_cnt))>>>( d_r2_str, d_r1_dict_value, sampleTimes, r1_cnt, r2_cnt, r1_dict_cnt, d_r1_dict_keys, sampleStrides, threadCount,i);

		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &elapsedTime, start, stop );
		hipEventDestroy( start );
		hipEventDestroy( stop );
    
		err = hipGetLastError();
		if(err != hipSuccess){
			fprintf(stderr,"Failed to launch deviceDDI kernel device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}
		
		err = hipMemcpy(d_r1_dict_keys,r1_dict_keys,sizeof(Node)*r1_dict_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}

		err = hipMemcpy(r1_dict_value,d_r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyDeviceToHost);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_value from device to Host(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}

		err = hipMemcpy(d_r1_dict_value, r1_dict_value,sizeof(int)*(sampleTimes+1)*r1_dict_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r1_dict_value from host to device(error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE); 
		}

		err = hipMemcpy(d_r2_str,temp1,1000*sizeof(char)*r2_cnt,hipMemcpyHostToDevice);
		if(err != hipSuccess){
			fprintf(stderr,"Failed to copy device d_r2_str (error code %s) !\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);	
		}
	}
	printf("Sampling completed\n");
    err = hipFree(d_r1_dict_keys);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
    err = hipFree(d_r2_str);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r2_str (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
	free(temp1);
    //Allocate Z-score array pinned memory for optimized usage of memory transfer bandwidth
    err = hipHostMalloc((void**)&z_score_arr, sizeof(float)*r1_dict_cnt, hipHostMallocDefault);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate z-score host (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
	//z_score_arr = (float*)malloc(sizeof(float)*r1_dict_cnt);
	float * d_z_score_p;
    err = hipMalloc((void **)&d_z_score_p,sizeof(float)*r1_dict_cnt);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate device d_z_score_p (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }


    //Allocate P-value array pinned memory for optimized usage of memory transfer bandwidth
    err = hipHostMalloc((void**)&p_value_arr, sizeof(float)*r1_dict_cnt, hipHostMallocDefault);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate p-value host (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipMalloc((void **)&d_p_value,sizeof(float)*r1_dict_cnt);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to allocate device d_p_value (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }

    err = hipFree(d_r1_dict_keys);
    if(err != hipSuccess){
        fprintf(stderr,"Failed to free from device d_r1_dict_keys (error code %s) !\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
	printf("Z-score, P-value calculation for proteins begin\n");
	printf("Kernel deviceZP launch with %d blocks of %d threads each\n", (int)ceil(r1_dict_cnt/256.0),256);
    totalTime += elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    deviceZP<<<ceil(r1_dict_cnt/256.0), 256>>>(d_r1_dict_value, sampleTimes, r1_dict_cnt, d_z_score_p, d_p_value);

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &elapsedTime, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	err = hipGetLastError();
	if(err != hipSuccess){
		fprintf(stderr,"Failed to launch deviceZP kernel device(error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	printf("Z-score, P-value calculation for proteins completed\n");
	printf( "\n******** Total Running Time of Kernel = %0.5f seconds ******* \n", (elapsedTime+totalTime)/1000);
	printf("Copy output data to host memory\n");
	err = hipMemcpy(p_value_arr,d_p_value,sizeof(float)*r1_dict_cnt,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy from p-value device to host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	err = hipFree(d_r1_dict_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	err = hipFree(d_p_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_p_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	//float *d_z_score;
    //float *z_score_arr1 = (float*)malloc(sizeof(float)*r1_dict_cnt);;

	err = hipMemcpy(z_score_arr,d_z_score_p,sizeof(float)*r1_dict_cnt,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to copy from z-score device to host (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
	
	err = hipFree(d_z_score_p);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free from device d_z_score (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	//Initialize sortedIndex2, this will hold correct index of the dictionary 1 records after sorting based on descending order of Z-score
	int sortedIndex2[r1_dict_cnt];
	for(i=0;i<r1_dict_cnt;i++){
		sortedIndex2[i] = i;
	}
	//Sort the array based on descending order of Z-score
	quickSort(z_score_arr, sortedIndex2, 0, r1_dict_cnt-1);
	//Write to output files
	strcpy(filename1, argv[6]);
	strcat(filename1, "_temp_result1_Substance_proteins_cutoff_");
	strcat(filename1,cutoffstr);
	strcat(filename1,"_p_");
	strcat(filename1,pvaluestr);
	strcpy(filename2, argv[6]);
	strcat(filename2, "_temp_result1_Substance_proteins_cutoff_");
	strcat(filename2,cutoffstr);
	strcat(filename2,".txt");

	printf("Write With star output files\n");
	op1 = fopen(filename1, "w");
	fprintf(op1,"Term Pair\tMeSHID\tDistribution\tZ-Score\tP-value\n");
	op2 = fopen(filename2, "w");
	fprintf(op2,"Term Pair\tMeSHID\tDistribution\tZ-Score\tP-value\n");
	k=0;
	for(i=0;i<r1_dict_cnt;i++){
		//char key[80];
		//j=0;
		/*while(r1_dict_list[k] != '~'){
			key[j++] = r1_dict_list[k++];
		}*/
		//key[j] = '\0';
		if((r1_dict_value[(sortedIndex2[i]*(sampleTimes+1))]>=cutoff) && (p_value_arr[sortedIndex2[i]]<=p_value)){
			fprintf(op1,"%s;%s\t[",argv[6],r1_dict_keys[i].key);
			for(j=0;j<sampleTimes;j++)
				fprintf(op1,"%d, ",r1_dict_value[(sortedIndex2[i]*(sampleTimes+1))+j]);
			fprintf(op1,"%d]\t%f\t%f\n",r1_dict_value[(sortedIndex2[i]*(sampleTimes+1))+j], z_score_arr[i],p_value_arr[sortedIndex2[i]]);
		}
		if((r1_dict_value[(sortedIndex2[i]*(sampleTimes+1))]>=cutoff) && (p_value_arr[sortedIndex2[i]]<=1.0)){
			fprintf(op2,"%s;%s\t[",argv[6],r1_dict_keys[i].key);
			for(j=0;j<=sampleTimes;j++)
				fprintf(op2,"%d, ",r1_dict_value[(sortedIndex2[i]*(sampleTimes+1))+j]);
			fprintf(op2,"%d]\t%f\t%f\n",r1_dict_value[(sortedIndex2[i]*(sampleTimes+1))+j], z_score_arr[i],p_value_arr[sortedIndex2[i]]);
		}
		k++;
	}
	fclose(op1);
	fclose(op2);

	printf("Processing completed\n");
	//free(r1_dict_list);
	free(r1_dict_keys);
	//free(temp1);
	err = hipHostFree(p_value_arr);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host p_value_arr (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipHostFree(z_score_arr);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host z_score_arr (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}

	err = hipHostFree(r1_dict_value);
	if(err != hipSuccess){
		fprintf(stderr,"Failed to free pinned host r1_dict_value (error code %s) !\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);	
	}
}